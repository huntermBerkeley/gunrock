#include <gunrock/algorithms/bfs.hxx>
#include <gunrock/algorithms/dawn.hxx>
#include <gunrock/util/performance.hxx>
#include <gunrock/io/parameters.hxx>
#include <gunrock/framework/benchmark.hxx>

#include <caching/pointers/set_cache.cuh>

#include "bfs_cpu.hxx"  // Reference implementation

using namespace gunrock;
using namespace memory;

void test_bfs_dev(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // std::string DEFAULT_BFS_ALGORITHMS =
  //     "DAWN";  // Using 'Breadth First Search' here will call the original BFS
  // --
  // IO

  std::string DEFAULT_BFS_ALGORITHMS = "Breadth First Search";
      

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        DEFAULT_BFS_ALGORITHMS);

  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load_large(params.filename);

  csr_t csr;

  if (params.binary) {

    printf("Reading binary\n");
    csr.read_binary(params.filename);
  } else {

    printf("Reading from COO\n");

    csr.from_coo_large(coo);

    std::string binary_name = params.filename+".bin";

    printf("Writing binary file %s\n", binary_name.c_str());

    csr.write_binary(binary_name);

  }



  // --
  // Build graph

  printf("CSR BUILT\n");

  auto G = graph::build<memory_space_t::device>(properties, csr);

  printf("Graph built\n");

  // --
  // Params and memory allocation

  size_t n_vertices = G.get_number_of_vertices();
  size_t n_edges = G.get_number_of_edges();
  thrust::device_vector<vertex_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);

  // Parse sources
  std::vector<int> source_vect;
  gunrock::io::cli::parse_source_string(params.source_string, &source_vect,
                                        n_vertices, params.num_runs);
  // Parse tags
  std::vector<std::string> tag_vect;
  gunrock::io::cli::parse_tag_string(params.tag_string, &tag_vect);

  // --
  // Run problem

  size_t n_runs = source_vect.size();
  std::vector<float> run_times;

  auto benchmark_metrics = std::vector<benchmark::host_benchmark_t>(n_runs);
  for (int i = 0; i < n_runs; i++) {
    benchmark::INIT_BENCH();
    if (DEFAULT_BFS_ALGORITHMS == "DAWN")
      run_times.push_back(gunrock::dawn_bfs::run(G, source_vect[i],
                                                 distances.data().get(),
                                                 predecessors.data().get()));
    else
      run_times.push_back(gunrock::bfs::run(G, source_vect[i],
                                            distances.data().get(),
                                            predecessors.data().get()));

    benchmark::host_benchmark_t metrics = benchmark::EXTRACT();
    benchmark_metrics[i] = metrics;

    benchmark::DESTROY_BENCH();
  }

  // Export metrics
  if (params.export_metrics) {
    if (DEFAULT_BFS_ALGORITHMS == "DAWN")
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "dawn_bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
    else
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
  }

  // Print info for last run
  std::cout << "Source : " << source_vect.back() << "\n";
  print::head(distances, 40, "GPU distances");
  std::cout << "GPU Elapsed Time : " << run_times[n_runs - 1] << " (ms)"
            << std::endl;

  // --
  // CPU Run

  if (params.validate) {
    thrust::host_vector<vertex_t> h_distances(n_vertices);
    thrust::host_vector<vertex_t> h_predecessors(n_vertices);

    // Validate with last source in source vector
    float cpu_elapsed = bfs_cpu::run<csr_t, vertex_t, edge_t>(
        csr, source_vect.back(), h_distances.data(), h_predecessors.data());

    int n_errors =
        util::compare(distances.data().get(), h_distances.data(), n_vertices);
    print::head(h_distances, 40, "CPU Distances");

    std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
    std::cout << "Number of errors : " << n_errors << std::endl;
  }
}

void test_bfs_host(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = uint64_t;
  using edge_t = uint64_t;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::host, vertex_t, edge_t, weight_t>;

  std::string DEFAULT_BFS_ALGORITHMS =
      "DAWN";  // Using 'Breadth First Search' here will call the original BFS
  // --
  // IO

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        DEFAULT_BFS_ALGORITHMS);


  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load_large(params.filename);

  csr_t csr;

  if (params.binary) {

    printf("Reading binary\n");
    csr.read_binary(params.filename);
  } else {

    printf("Reading from COO\n");

    csr.from_coo_large(coo);

    std::string binary_name = params.filename+".bin";

    printf("Writing binary file %s\n", binary_name.c_str());

    csr.write_binary(binary_name);
    
  }

  // --
  // Build graph

  printf("CSR BUILT\n");

  auto G = graph::build<memory_space_t::host>(properties, csr);


  printf("GRAPH BUILT %lu vert\n", G.get_number_of_vertices());
  // --
  // Params and memory allocation

  size_t n_vertices = G.get_number_of_vertices();
  size_t n_edges = G.get_number_of_edges();
  // thrust::device_vector<vertex_t> distances(n_vertices);
  // thrust::device_vector<vertex_t> predecessors(n_vertices);

  // thrust::host_vector<vertex_t> distances(n_vertices);
  // thrust::host_vector<vertex_t> predecessors(n_vertices);


  vertex_t * distances;

  hipHostMalloc((void **)&distances, sizeof(vertex_t)*n_vertices);

  vertex_t * predecessors;

  hipHostMalloc((void **)&predecessors, sizeof(vertex_t)*n_vertices);


  printf("Thrust memory alloced\n");

  // Parse sources
  std::vector<int> source_vect;
  gunrock::io::cli::parse_source_string(params.source_string, &source_vect,
                                        n_vertices, params.num_runs);
  // Parse tags
  std::vector<std::string> tag_vect;
  gunrock::io::cli::parse_tag_string(params.tag_string, &tag_vect);

  // --
  // Run problem



  size_t n_runs = source_vect.size();
  std::vector<float> run_times;

  auto benchmark_metrics = std::vector<benchmark::host_benchmark_t>(n_runs);
  for (int i = 0; i < n_runs; i++) {

    vertex_t source = source_vect[i];
    benchmark::INIT_BENCH();
    if (DEFAULT_BFS_ALGORITHMS == "DAWN")
      run_times.push_back(gunrock::dawn_bfs::run(G, source,
                                                 distances,
                                                 predecessors));
    else
      run_times.push_back(gunrock::bfs::run(G, source,
                                            distances,
                                            predecessors));

    benchmark::host_benchmark_t metrics = benchmark::EXTRACT();
    benchmark_metrics[i] = metrics;

    benchmark::DESTROY_BENCH();
  }

  // Export metrics
  if (params.export_metrics) {
    if (DEFAULT_BFS_ALGORITHMS == "DAWN")
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "dawn_bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
    else
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
  }

  // Print info for last run
  std::cout << "Source : " << source_vect.back() << "\n";
  //print::head(distances, 40, "GPU distances");
  std::cout << "GPU Elapsed Time : " << run_times[n_runs - 1] << " (ms)"
            << std::endl;

  // --
  // CPU Run

  // if (params.validate) {
  //   thrust::host_vector<vertex_t> h_distances(n_vertices);
  //   thrust::host_vector<vertex_t> h_predecessors(n_vertices);

  //   // Validate with last source in source vector
  //   float cpu_elapsed = bfs_cpu::run<csr_t, vertex_t, edge_t>(
  //       csr, source_vect.back(), h_distances.data(), h_predecessors.data());

  //   int n_errors =
  //       util::compare(distances, h_distances.data(), n_vertices);
  //   print::head(h_distances, 40, "CPU Distances");

  //   std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  //   std::cout << "Number of errors : " << n_errors << std::endl;
  // }
}

int main(int argc, char** argv) {
  //test_bfs_dev(argc, argv);
  test_bfs_host(argc, argv);
}
