#include <gunrock/algorithms/sssp.hxx>
#include <gunrock/algorithms/dawn.hxx>
#include <gunrock/util/performance.hxx>
#include <gunrock/io/parameters.hxx>
#include <gunrock/framework/benchmark.hxx>

#include <caching/pointers/set_cache.cuh>

#include "bfs_cpu.hxx"  // Reference implementation

using namespace gunrock;
using namespace memory;


bool fileExists(const std::string& filename) {
  std::ifstream file(filename);
  bool exists = file.good();
  return exists;
}

template<memory_space_t space,
          typename index_t,
          typename offset_t,
          typename value_t>
format::csr_t<space, index_t, offset_t, value_t> load_graph(gunrock::io::cli::parameters_t params, format::coo_no_vector<space, index_t, offset_t, value_t> coo){

  format::csr_t<space, index_t, offset_t, value_t> csr;

  std::string filename = params.filename;

  if (fileExists(filename+".csr")){

    printf("Reading CSR binary\n");

    csr.read_in_csr(filename+".csr");

  } else {

    printf("converting COO\n");

    csr.from_coo_large(coo);

    std::string binary_name = params.filename+".csr";

    printf("Writing csr file %s\n", binary_name.c_str());

    csr.write_out_csr(binary_name);

  }

  return csr;

}


template<memory_space_t space,
          typename index_t,
          typename offset_t,
          typename value_t>
format::csr_t<space, index_t, offset_t, value_t> load_csr(gunrock::io::cli::parameters_t params){

  format::csr_t<space, index_t, offset_t, value_t> csr;

  std::string filename = params.filename;
  
  csr.read_in_csr(filename+".csr");

  return csr;

}


bool has_csr(gunrock::io::cli::parameters_t params){

  return fileExists(params.filename+".csr");

}

void test_cache_sssp(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = uint64_t;
  using edge_t = uint64_t;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::host, vertex_t, edge_t, weight_t>;

  // auto coo_t = 
  //     format::coo_no_vector<memory_space_t::host, vertex_t, edge_t, weight_t>;

  std::string DEFAULT_SSSP_ALGORITHMS =
      "DAWN";  // Using 'Breadth First Search' here will call the original BFS
  // --
  // IO


      if (has_csr){

      }

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        DEFAULT_SSSP_ALGORITHMS);


  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;



  gunrock::graph::graph_properties_t properties;

  csr_t csr;

  if (has_csr(params)){


    properties = mm.load_properties(params.filename);

    csr = load_csr<memory_space_t::host, vertex_t, edge_t, weight_t>(params);


  } else {

    auto [ext_properties, coo] = mm.load_large(params.filename);

    properties = ext_properties;
    csr = load_graph<memory_space_t::host, vertex_t, edge_t, weight_t>(params, coo);

  }


  // auto csr_properties = mm.load_properties(params.filename);
  // auto [properties, coo] = mm.load_large(params.filename);


  // auto [properties, coo] = mm.load_large(params.filename);

  // csr_t csr = load_graph<memory_space_t::host, vertex_t, edge_t, weight_t>(params, coo);

  // if (params.filename) {

  //   printf("Reading binary\n");
  //   csr.read_binary(params.filename);
  // } else {

  //   printf("Reading from COO\n");

  //   csr.from_coo_large(coo);

  //   std::string binary_name = params.filename+".csr";

  //   printf("Writing csr file %s\n", binary_name.c_str());

  //   csr.write_binary(binary_name);
    
  // }

  // --
  // Build graph

  printf("CSR BUILT\n");

  auto G = graph::build<memory_space_t::host>(properties, csr);


  printf("GRAPH BUILT %lu vert\n", G.get_number_of_vertices());
  // --
  // Params and memory allocation

  size_t n_vertices = G.get_number_of_vertices();
  size_t n_edges = G.get_number_of_edges();
  // thrust::device_vector<vertex_t> distances(n_vertices);
  // thrust::device_vector<vertex_t> predecessors(n_vertices);

  // thrust::host_vector<vertex_t> distances(n_vertices);
  // thrust::host_vector<vertex_t> predecessors(n_vertices);


  weight_t * distances;

  hipHostMalloc((void **)&distances, sizeof(weight_t)*n_vertices);

  vertex_t * predecessors;

  hipHostMalloc((void **)&predecessors, sizeof(vertex_t)*n_vertices);

  thrust::device_vector<int> edges_visited(1);
  thrust::device_vector<int> vertices_visited(1);

  printf("Thrust memory alloced\n");

  // Parse sources
  std::vector<int> source_vect;
  gunrock::io::cli::parse_source_string(params.source_string, &source_vect,
                                        n_vertices, params.num_runs);
  // Parse tags
  std::vector<std::string> tag_vect;
  gunrock::io::cli::parse_tag_string(params.tag_string, &tag_vect);

  // --
  // Run problem


  printf("Mem allocated, starting execution\n");

  size_t n_runs = source_vect.size();
  std::vector<float> run_times;


  auto benchmark_metrics = std::vector<benchmark::host_benchmark_t>(n_runs);
  for (int i = 0; i < n_runs; i++) {

    vertex_t source = source_vect[i];
    benchmark::INIT_BENCH();
    if (DEFAULT_SSSP_ALGORITHMS == "DAWN")
      run_times.push_back(gunrock::dawn_sssp::run(G, source,
                                                  distances,
                                                  predecessors));
    else
      run_times.push_back(gunrock::sssp::run(G, source,
                                             distances,
                                             predecessors));

    benchmark::host_benchmark_t metrics = benchmark::EXTRACT();
    benchmark_metrics[i] = metrics;

    benchmark::DESTROY_BENCH();
  }

  // Export metrics
  if (params.export_metrics) {
    if (DEFAULT_SSSP_ALGORITHMS == "DAWN")
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "dawn_bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
    else
      gunrock::util::stats::export_performance_stats(
          benchmark_metrics, n_edges, n_vertices, run_times, "bfs",
          params.filename, "market", params.json_dir, params.json_file,
          source_vect, tag_vect, num_arguments, argument_array);
  }

  //output.
  //print::head(distances, 40, "GPU distances");
  std::cout << "GPU Elapsed Time : " << run_times[params.num_runs - 1]
            << " (ms)" << std::endl;

  // --
  // CPU Run

  // if (params.validate) {
  //   thrust::host_vector<vertex_t> h_distances(n_vertices);
  //   thrust::host_vector<vertex_t> h_predecessors(n_vertices);

  //   // Validate with last source in source vector
  //   float cpu_elapsed = bfs_cpu::run<csr_t, vertex_t, edge_t>(
  //       csr, source_vect.back(), h_distances.data(), h_predecessors.data());

  //   int n_errors =
  //       util::compare(distances, h_distances.data(), n_vertices);
  //   print::head(h_distances, 40, "CPU Distances");

  //   std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  //   std::cout << "Number of errors : " << n_errors << std::endl;
  // }
}

int main(int argc, char** argv) {
  //test_bfs_dev(argc, argv);
  test_cache_sssp(argc, argv);
}
